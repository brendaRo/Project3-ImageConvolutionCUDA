#include "hip/hip_runtime.h"
#include "Image.h"
#include "PPM.h"
#include <iostream>
#include <cstdlib>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
using namespace std;



#define TILE_WIDTH 16
#define filcols 3
#define filrows 3
#define w (TILE_WIDTH + filcols -1)


__global__ void Convolution(float * InputImage, const float *__restrict__ filtro,
		float* new_img, int channels, int width, int height){

	__shared__ float BlockS[w][w];  						//block of image in shared memory


	// allocation in shared memory of image blocks
	int radio = filrows/2;
 	for (int k = 0; k < channels; k++) {
 		int dest = threadIdx.y * TILE_WIDTH + threadIdx.x;
 		int RY = dest/w;    						//row of shared memory
 		int CX = dest%w;						//col of shared memory
 		int srcY = blockIdx.y *TILE_WIDTH + RY - radio; 		//fetch the data from input image
 		int srcX = blockIdx.x *TILE_WIDTH + RX- radio;	
 		int src = (srcY *width +srcX) * channels + k;   		// input image
 		if(srcY>= 0 && srcY < height && srcX>=0 && srcX < width)
 			BlockS[RY][RX] = InputImage[src];  			// copy element of image in shared memory
 		else
 			BlockS[RY][RX] = 0;



 		dest = threadIdx.y * TILE_WIDTH+ threadIdx.x + TILE_WIDTH * TILE_WIDTH;
 		RY = dest/w;
		RX = dest%w;
		srcY = blockIdx.y *TILE_WIDTH + RY - radio;
		srcX = blockIdx.x *TILE_WIDTH + RX - radio;
		src = (srcY *width +srcX) * channels + k;
		if(RY < w){
			if(srcY>= 0 && srcY < height && srcX>=0 && srcX < width)
				BlockS[RY][RX] = InputImage[src];
			else
				BlockS[RY][RX] = 0;
		}

 		__syncthreads();


 		//compute filter and image convolution
		
 		float accum = 0;
 		int y, x;

 		for (y= 0; y < filcols; y++)
 			for(x = 0; x<filrows; x++)
 				accum += BlockS[threadIdx.y + y][threadIdx.x + x] *filtro[y * filcols + x];

 		y = blockIdx.y * TILE_WIDTH + threadIdx.y;
 		x = blockIdx.x * TILE_WIDTH + threadIdx.x;
 		if(y < height && x < width)
 			new_img[(y * width + x) * channels + k] = accum;
 		__syncthreads();
 	}

}



int main(int argc, char **argv){

	char *iter = argv[1];
  	char *imgpath = argv[2];
	char *filterpath = argv[3];
	char *imgrespath = argv[4];

	//1. Set memory for variables in host and device
	int imgCh;
	int imgH;
	int imgW;
	Image_t* inputImage;
	Image_t* outputImage;
	float* hostInputImage;
	float* hostOutputImage;
	float* deviceInputImage;
	float* deviceOutputImage;
	float* devicefilter;
	float filter[filrows * filcols];
	
	//time variables
	hipEvent_t start, stop;
	float t;

	if(imgpath == "lena"){
		inputImage = PPM_import("lena.ppm");
	} else if (imgpath == "buildings"){
		inputImage = PPM_import("edificios1.ppm");
	} else if (imgpath == "landscape"){
		inputImage = PPM_import("paisaje1.ppm");
	}

	imgW = Image_getWidth(inputImage);
	imgH = Image_getHeight(inputImage);
	imgCh = Image_getChannels(inputImage);

	outputImage = Image_new(imgW, imgH, imgCh);

	hostInputImage = Image_getData(inputImage);
	hostOutputImage = Image_getData(outputImage);

	
	//2. Copy data from Host to Device
	hipMalloc((void **) &deviceInputImage, imgW * imgH *
			imgCh * sizeof(float));
	hipMalloc((void **) &deviceOutputImage, imgW * imgH *
			imgCh * sizeof(float));
	hipMalloc((void **) &devicefilter, filrows * filcols
			* sizeof(float));
	hipMemcpy(deviceInputImage, hostInputImage,
			imgW * imgH * imgCh * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(devicefilter, filter,
			filrows * filcols * sizeof(float),
			hipMemcpyHostToDevice);


	//Grid dimensions block and grid
	dim3 dimGrid(ceil((float) imgW/TILE_WIDTH),
			ceil((float) imgH/TILE_WIDTH));
	dim3 dimBlock(TILE_WIDTH,TILE_WIDTH,1);

	// 3. execute convolution
	hipEventCreate(&start); hipEventCreate(&stop); 
	hipEventRecord(start, 0);

	Convolution<<<dimGrid,dimBlock>>>(deviceInputImage, devicefilter, deviceOutputImage,
	imgCh, imgW, imgH);
	hipEventRecord(stop, 0);

	hipEventElapsedTime(&t, start, stop);                                                                                                                                                                 
	hipEventDestroy(start); hipEventDestroy(stop);
	//Tiempo
	printf("convolution time:%f ms",t/1000.0);


	//4. Copy data from Device to Host
	hipMemcpy(hostOutputImage, deviceOutputImage, imgW * imgH *
			imgCh * sizeof(float), hipMemcpyDeviceToHost);

	PPM_export("result.ppm", outputImage);

	
	//5. Free memories
	hipFree(deviceInputImage);
	hipFree(deviceOutputImage);
	hipFree(devicefilter);

	Image_delete(outputImage);
	Image_delete(inputImage);


}